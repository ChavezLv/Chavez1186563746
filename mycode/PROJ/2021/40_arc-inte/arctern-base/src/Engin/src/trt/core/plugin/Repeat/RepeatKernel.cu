#include "hip/hip_runtime.h"
//
// Created by gpu on 19-12-2.
//
#include "RepeatKernel.h"

template <typename DType>
__global__ void RepeatKernel_axis2(DType* output, const DType* input,
                                   const int input_shape_c, const int input_shape_h, const int input_shape_w,
                                   const int output_shape_c, const int output_shape_h, const int output_shape_w,
                                   int repeat){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;
  if(index >= output_shape_h * output_shape_w){
    return;
  }

  int dst_x = index % output_shape_w;
  int dst_y = index / output_shape_w;

  int src_x = dst_x;
  int src_y = dst_y / repeat;

  DType src_value = input[batch * input_shape_c * input_shape_h * input_shape_w +
                          plane * input_shape_h * input_shape_w + src_y * input_shape_w + src_x];
  output[batch * output_shape_c * output_shape_h * output_shape_w + plane * output_shape_h * output_shape_w +
         dst_y * output_shape_w + dst_x] = src_value;
}

template <typename DType>
__global__ void RepeatKernel_axis3(DType* output, const DType* input,
                                   const int input_shape_c, const int input_shape_h, const int input_shape_w,
                                   const int output_shape_c, const int output_shape_h, const int output_shape_w,
                                   int repeat){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;
  if(index >= output_shape_h * output_shape_w){
    return;
  }

  int dst_x = index % output_shape_w;
  int dst_y = index / output_shape_w;

  int src_x = dst_x / repeat;
  int src_y = dst_y;

  DType src_value = input[batch * input_shape_c * input_shape_h * input_shape_w +
                          plane * input_shape_h * input_shape_w + src_y * input_shape_w + src_x];
  output[batch * output_shape_c * output_shape_h * output_shape_w + plane * output_shape_h * output_shape_w +
         dst_y * output_shape_w + dst_x] = src_value;
}

pluginStatus_t RepeatInference(hipStream_t stream, void* outputs_0, const void* inputs_0,
                               int batchsize, const int* input_shape, const int* output_shape,
                               int repeats, int axis){
  const int kBaseThreadBits = 8;
  const int kBaseThreadNum = 1 << kBaseThreadBits;
  int xGridSize = (output_shape[2] * output_shape[3] + kBaseThreadNum - 1) / kBaseThreadNum;
  dim3 dimGrid(xGridSize, output_shape[1], output_shape[0] * batchsize);

  if(axis == 2){
    RepeatKernel_axis2<float><<<dimGrid, kBaseThreadNum, 0, stream>>>((float*)outputs_0, (float*)inputs_0, input_shape[1],
            input_shape[2], input_shape[3], output_shape[1], output_shape[2], output_shape[3], repeats);
  }else if(axis == 3){
    RepeatKernel_axis3<float><<<dimGrid, kBaseThreadNum, 0, stream>>>((float*)outputs_0, (float*)inputs_0, input_shape[1],
            input_shape[2], input_shape[3], output_shape[1], output_shape[2], output_shape[3], repeats);
  }

  return STATUS_SUCCESS;
}