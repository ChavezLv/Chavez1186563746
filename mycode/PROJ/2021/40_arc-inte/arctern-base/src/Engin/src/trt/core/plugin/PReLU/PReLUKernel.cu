#include "hip/hip_runtime.h"
#include "PReLUKernel.h"

template <typename DType>
__global__ void PReluKernel(const int num_threads, DType* out, const DType* in_data, const DType* in_alpha,
    const int channel_stride, const int alpha_stride)
{
    int x0 = blockIdx.x * blockDim.x + threadIdx.x;
    for (int index = x0; index < num_threads; index += gridDim.x * blockDim.x)
    {
        int c = (index / channel_stride) % alpha_stride;
        out[index] = in_data[index] > 0 ? in_data[index] : in_data[index] * in_alpha[c];
    }
}

pluginStatus_t PReLUInference(hipStream_t stream, const int num_threads, void* outputs, const void* inputs_data,
    const void* inputs_alpha, const int ndim, const int* input_data_shape)
{
    const int kMaxGridNum = 65535;
    const int kBaseThreadBits = 8;
    const int kBaseThreadNum = 1 << kBaseThreadBits;

    int ngrid = std::min(kMaxGridNum, (num_threads + kBaseThreadNum - 1) / kBaseThreadNum);

    if (ndim == 2)
    {
        PReluKernel<float><<<ngrid, kBaseThreadNum, 0, stream>>>(
            num_threads, (float*) outputs, (float*) inputs_data, (float*) inputs_alpha, 1, input_data_shape[1]);
    }
    else if (ndim == 3)
    {
        PReluKernel<float><<<ngrid, kBaseThreadNum, 0, stream>>>(num_threads, (float*) outputs, (float*) inputs_data,
            (float*) inputs_alpha, input_data_shape[2], input_data_shape[1]);
    }
    else if (ndim == 4)
    {
        PReluKernel<float><<<ngrid, kBaseThreadNum, 0, stream>>>(num_threads, (float*) outputs, (float*) inputs_data,
            (float*) inputs_alpha, input_data_shape[2] * input_data_shape[3], input_data_shape[1]);
    }

    return STATUS_SUCCESS;
}
