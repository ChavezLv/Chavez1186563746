#include "hip/hip_runtime.h"
//
// Created by gpu on 19-12-2.
//
#include "ErfKernel.h"
#include "hip/hip_runtime.h"

template <typename DType>
__global__ void ErfKernel(const int threadNum, DType* out, const DType* input){
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < threadNum)
  {
    out[index] = erf(input[index]);
  }
}

pluginStatus_t ErfInference(hipStream_t stream, const int threadNum, void* outputs, const void* inputs){
  const int kMaxGridNum = 65536;
  dim3 block(256);
  dim3 grid(std::min(kMaxGridNum, (int)((threadNum + block.x -1) / block.x)));
  ErfKernel<float><<<grid, block, 0, stream>>>(threadNum, (float*)outputs, (float*)inputs);
  return STATUS_SUCCESS;
}