#include "hip/hip_runtime.h"
//
// Created by gpu on 19-12-2.
//
#include "ModSaclarKernal.h"

template <typename DType>
__global__ void ModSaclarKernel(const int threadNum, DType* out, const DType* input, int scalar){
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < threadNum)
  {
    out[index] = (int)input[index] % scalar;
  }
}

template <typename DType>
__global__ void DivSaclarKernel(const int threadNum, DType* out, const DType* input, int scalar){
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < threadNum)
  {
    out[index] = input[index] / scalar;
  }
}

pluginStatus_t ModSaclarInference(hipStream_t stream, const int& threadNum, void* outputs, const void* inputs,
                                  int& scalar, int& mode){
  int kMaxGridNum = 65536;
  dim3 block(256);
  dim3 grid(std::min(kMaxGridNum, (int)((threadNum + block.x -1) / block.x)));
  switch (mode){
    case 0:/// mod op pass to element
        ModSaclarKernel<float><<<grid, block, 0, stream>>>(threadNum, (float*)outputs, (float*)inputs, scalar);
        break;
    case 1: /// div op pass to element
        DivSaclarKernel<float><<<grid, block, 0, stream>>>(threadNum, (float*)outputs, (float*)inputs, scalar);
        break;
    default: break;
  }

  return STATUS_SUCCESS;
}
