#include "hip/hip_runtime.h"
//
// Created by gpu on 19-12-2.
//
#include "EqualKernel.h"

template <typename DType>
__global__ void EqualKernel(const int threadNum, DType* out, const DType* inputs_0, const DType* inputs_1){
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < threadNum)
  {
    out[index] = (inputs_0[index] == inputs_1[index]) ? 1 : 0;
  }
}

pluginStatus_t EqualInference(hipStream_t stream, const int threadNum, void* outputs, const void* inputs_0,
                              const void* inputs_1){
  const int kMaxGridNum = 65536;
  dim3 block(256);
  dim3 grid(std::min(kMaxGridNum, (int)((threadNum + block.x -1) / block.x)));
  EqualKernel<float><<<grid, block, 0, stream>>>(threadNum, (float*)outputs, (float*)inputs_0, (float*)inputs_1);
  return STATUS_SUCCESS;
}