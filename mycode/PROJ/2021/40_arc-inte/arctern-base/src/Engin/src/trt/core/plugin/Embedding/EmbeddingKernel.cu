#include "hip/hip_runtime.h"
#include "EmbeddingKernel.h"

template <typename DType, typename IType, bool clip = true>
__global__ void EmbeddingKernel(
    const int num_threads, DType* out_data, const DType* in_data, const IType* idx, const int M, const int K)
{
    int x0 = blockIdx.x * blockDim.x + threadIdx.x;

    for (int index = x0; index < num_threads; index += gridDim.x * blockDim.x)
    {
        int j = idx[index / M];
        if (clip)
        {
            if (j <= 0)
                j = 0;
            else if (j >= K)
                j = K - 1;
        }
        else
        {
            j = j % K;
            j += (j < 0) ? K : 0;
        }
        out_data[index] = in_data[j * M + index % M];
    }
}

pluginStatus_t EmbeddingInference(hipStream_t stream, const int num_threads, void* outputs, const void* inputs_data,
    const void* idx, const int output_dim, const int input_dim)
{
    const int kMaxGridNum = 65535;
    const int kBaseThreadBits = 8;
    const int kBaseThreadNum = 1 << kBaseThreadBits;

    int ngrid = std::min(kMaxGridNum, (num_threads + kBaseThreadNum - 1) / kBaseThreadNum);

    EmbeddingKernel<float><<<ngrid, kBaseThreadNum, 0, stream>>>(
        num_threads, (float*) outputs, (float*) inputs_data, (float*) idx, output_dim, input_dim);

    return STATUS_SUCCESS;
}
