#include "hip/hip_runtime.h"
#include "../cub/block/block_reduce.cuh"

#include "L2NormalizationKernel.h"

template <typename DType, int BLOCK_THREADS>
__global__ void L2NormalizationKernel(
    DType* out, const DType* in, const int batch_size, const int batch_step, const float eps)
{
    typedef hipcub::BlockReduce<float, BLOCK_THREADS> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int x0 = threadIdx.x;
    int batch0 = blockIdx.x;

    for (int batch = batch0; batch < batch_size; batch += gridDim.x)
    {
        int offset = batch * batch_step;

        float sum = 0.0f;
        __shared__ float norm;

        for (int index = x0; index < batch_step; index += blockDim.x)
        {
            const float data = in[offset + index];
            sum += data * data;
        }
        float reduce_result = BlockReduce(temp_storage).Sum(sum);

        if (threadIdx.x == 0)
        {
            norm = sqrtf(reduce_result);
            norm = fmaxf(norm, eps);
        }
        __syncthreads();

        for (int index = x0; index < batch_step; index += blockDim.x)
        {
            out[offset + index] = in[offset + index] / norm;
        }
    }
}

pluginStatus_t L2NormalizationInference(hipStream_t stream, void* outputs, const void* inputs_data,
    const int batch_size, const int batch_step, const float eps)
{
    const int kMaxGridNum = 65535;
    const int kBaseThreadBits = 8;
    const int kBaseThreadNum = 1 << kBaseThreadBits;

    int ngrid = std::min(kMaxGridNum, batch_size);

    L2NormalizationKernel<float, kBaseThreadNum>
        <<<ngrid, kBaseThreadNum, 0, stream>>>((float*) outputs, (float*) inputs_data, batch_size, batch_step, eps);

    return STATUS_SUCCESS;
}
