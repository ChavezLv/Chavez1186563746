#include "hip/hip_runtime.h"
#include "LastDimTopKKernel.h"

#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>

constexpr int kWarpSize = 32;

/// The maximum in-block bitonic sort we support
constexpr int kMaxBitonicSortSize = 4096;

__device__ __forceinline__ int getLaneId()
{
    int laneId;
    asm("mov.s32 %0, %laneid;" : "=r"(laneId));
    return laneId;
}

__device__ __forceinline__ unsigned getLaneMaskLt()
{
    unsigned mask;
    asm("mov.u32 %0, %%lanemask_lt;" : "=r"(mask));
    return mask;
}

// Returns log2(n) for a positive integer type
template <typename T>
__device__ constexpr int IntegerLog2(T n, int p = 0)
{
    return (n <= 1) ? p : IntegerLog2(n / 2, p + 1);
}

// Returns true if the given integer type is a power-of-2 (positive only)
template <typename T>
__device__ constexpr bool integerIsPowerOf2(T v)
{
    return (v && !(v & (v - 1)));
}

template <typename T>
__device__ inline void swapVars(T& t1, T& t2)
{
    T tmp = t1;
    t1 = t2;
    t2 = tmp;
}

template <typename K, typename V>
struct LTComp
{
    __device__ inline bool operator()(const K& kA, const V& vA, const K& kB, const V& vB) const
    {
        // FIXME: adding value comparison is slow
        return (kA < kB) || ((kA == kB) && (vA < vB));
    }
};

template <typename K, typename V>
struct GTComp
{
    __device__ inline bool operator()(const K& kA, const V& vA, const K& kB, const V& vB) const
    {
        // FIXME: adding value comparison is slow
        // FIXME: it's vA < vB because the sorting order for V (aka
        // indices) is different in our use case
        return (kA > kB) || ((kA == kB) && (vA < vB));
    }
};

constexpr size_t getHeapSmemSize(size_t keySize, size_t valueSize, int numThreads, int heapSize)
{
    return (numThreads / kWarpSize) * heapSize * (keySize + valueSize);
}

template <typename Comparator, typename K, typename V>
__device__ inline void bitonicSwap(K& kA, V& vA, K& kB, V& vB, bool dir, const Comparator& comp)
{
    bool swap = comp(kA, vA, kB, vB);
    if (swap == dir)
    {
        swapVars(kA, kB);
        swapVars(vA, vB);
    }
};

template <typename Comparator, typename K, typename V, int Power2SortSize, int ThreadsPerBlock>
__device__ inline void bitonicSort(K* keys, V* values, const Comparator& comp)
{
    static_assert(Power2SortSize <= kMaxBitonicSortSize, "sort size <= 4096 only supported");
    // Assume the sort is taking place in shared memory
    // static_assert(Power2SortSize * (sizeof(K) + sizeof(V)) < 32768,
    //               "sort data too large (>32768 bytes)");
    static_assert(integerIsPowerOf2(Power2SortSize), "sort size must be power of 2");
    static_assert(integerIsPowerOf2(ThreadsPerBlock), "threads in block must be power of 2");

    // If what we are sorting is too small, then not all threads
    // participate
    constexpr int numThreadsForSort = Power2SortSize / 2;
    constexpr bool allThreads = numThreadsForSort >= ThreadsPerBlock;

    // If what we are sorting is too large, then threads must loop more
    // than once
    constexpr int loopPerThread = allThreads ? numThreadsForSort / ThreadsPerBlock : 1;

#pragma unroll
    for (int size = 2; size < Power2SortSize; size *= 2)
    {

#pragma unroll
        for (int stride = size / 2; stride > 0; stride /= 2)
        {

#pragma unroll
            for (int loop = 0; loop < loopPerThread; ++loop)
            {
                int threadId = loop * ThreadsPerBlock + threadIdx.x;
                bool flag = ((threadId & (size / 2)) != 0);

                int pos = 2 * threadId - (threadId & (stride - 1));

                if (allThreads || (threadId < numThreadsForSort))
                {
                    bitonicSwap<Comparator, K, V>(
                        keys[pos], values[pos], keys[pos + stride], values[pos + stride], flag, comp);
                }

                __syncthreads();
            }
        }
    }

#pragma unroll
    for (int stride = Power2SortSize / 2; stride > 0; stride /= 2)
    {

#pragma unroll
        for (int loop = 0; loop < loopPerThread; ++loop)
        {
            int threadId = loop * ThreadsPerBlock + threadIdx.x;

            int pos = 2 * threadId - (threadId & (stride - 1));

            if (allThreads || (threadId < numThreadsForSort))
            {
                bitonicSwap<Comparator, K, V>(
                    keys[pos], values[pos], keys[pos + stride], values[pos + stride], false, comp);
            }

            __syncthreads();
        }
    }
}

// Per-warp heap structure in shared memory:
// [key_0, ..., key_(HeapSize-2)], [empty element] (warp 0)
// ...
// [key_0, ..., key_(HeapSize-2)], [empty element] (warp n-1)
// [value_0, ..., value_(HeapSize-2)], [empty element] (warp 0)
// ...
// [value_0, ..., value_(HeapSize-2)], [empty element] (warp n-1)

// Dir == true means we are selecting the largest values, thus
// the heap is a min-heap
template <typename K, typename V, int HeapSize, bool Dir>
__device__ inline void warpHeapInsert(K k, V v, K* keyHeap, V* valueHeap)
{
    // Replace head if we are < head
    bool valid = Dir ? (k > keyHeap[0]) : (k < keyHeap[0]);

    // Even though this is the single-thread case, another preceding
    // thread in the warp may have inserted in a new element that
    // supersedes our element and thus our attempt at an insert would do
    // nothing.
    if (!valid)
    {
        return;
    }

    // Swap with head if valid
    K currentKey = k;
    V currentValue = v;

    keyHeap[0] = currentKey;
    valueHeap[0] = currentValue;

    // The number of interior nodes in the heap is log2(HeapSize / 2):
    // heap size 8 means there are 7 elements in the heap, indices 0-6
    // (0 12 3456)
    // log2(8 / 2) = 2 levels of interior nodes for heap size 8 (0 and 12)
    int i = 0;
#ifndef __HIP_PLATFORM_HCC__
#pragma unroll
#endif
    for (int levels = 0; levels < IntegerLog2(HeapSize / 2); ++levels)
    {
        int leftChild = i * 2 + 1;
        int rightChild = leftChild + 1;
        K leftKey = keyHeap[leftChild];
        K rightKey = keyHeap[rightChild];

        // What child might we want to swap with (max heap = larger child;
        // min heap = smaller child)
        bool swap = Dir ? (leftKey < rightKey) : (leftKey > rightKey);
        int childToSwap = swap ? leftChild : rightChild;
        K keyChildToSwap = swap ? leftKey : rightKey;

        // If we're bigger than both children (max heap), or smaller than
        // both children (min heap), then we do nothing for the rest of
        // the iterations
        valid = Dir ? !(currentKey < keyChildToSwap) : !(currentKey > keyChildToSwap);

        // Swap with childToSwap if still valid
        keyHeap[i] = valid ? keyChildToSwap : currentKey;
        valueHeap[i] = valid ? valueHeap[childToSwap] : currentValue;

        keyHeap[childToSwap] = valid ? currentKey : keyChildToSwap;
        valueHeap[childToSwap] = valid ? currentValue : valueHeap[childToSwap];

        i = childToSwap;

        // This is our new element to potentially downheap
        currentKey = keyHeap[i];
        currentValue = valueHeap[i];
    }
}

template <typename K, typename V, int HeapSize, bool Dir>
__device__ inline void warpHeap(K k, V v, K& keyHeapHead, K* keyHeap, V* valueHeap)
{
    // All threads in the warp have elements
    bool wantInsert = Dir ? (k > keyHeapHead) : (k < keyHeapHead);

    // Find out all the lanes that have elements to add to the heap
    unsigned int vote = __ballot_sync(__activemask(), wantInsert);

    if (!vote)
    {
        // Everything the warp has is smaller than our heap
        return;
    }

    // Otherwise, we want to serialize execution of the threads
    // that have elements
    int index = __popc(getLaneMaskLt() & vote);
    int total = __popc(vote);

    // FIXME: try switch statement and explicitly handle cases
    // FIXME: how do cases work?
    for (int i = 0; i < total; ++i)
    {
        if (index == i && wantInsert)
        {
            // Insert into our heap
            warpHeapInsert<K, V, HeapSize, Dir>(k, v, keyHeap, valueHeap);

            // Make sure all smem writes are visible
            __threadfence_block();
        }
    }

    // Re-broadcast the new heap head
    // FIXME: consider each updater above will broadcast its value with
    // a shuffle instead?
    keyHeapHead = keyHeap[0];
}

template <typename K, typename V, int ThreadsPerBlock, int HeapSize, bool Dir>
class Heap
{
public:
    static constexpr size_t getSmemSize()
    {
        return getHeapSmemSize(sizeof(K), sizeof(V), ThreadsPerBlock, HeapSize);
    }

    __device__ Heap(void* smem, K initKey, V initVal)
    {
        heapBase = smem;

        int warpId = threadIdx.x / kWarpSize;
        int laneId = getLaneId();

        auto kStart = getKeyStart();
        heapK = &kStart[warpId * HeapSize];
        auto vStart = getValueStart();
        heapV = &vStart[warpId * HeapSize];

        heapHead = initKey;

        if (HeapSize < kWarpSize)
        {
            if (laneId < HeapSize)
            {
                heapK[laneId] = initKey;
                heapV[laneId] = initVal;
            }
        }
        else
        {
#pragma unroll
            for (int i = 0; i < HeapSize / kWarpSize; ++i)
            {
                heapK[laneId + i * kWarpSize] = initKey;
                heapV[laneId + i * kWarpSize] = initVal;
            }
        }
    }

    // Returns a pointer to the start of our block-wide key storage
    inline __device__ K* getKeyStart()
    {
        return (K*) heapBase;
    }

    // Returns a pointer to the start of our block-wide value storage
    inline __device__ V* getValueStart()
    {
        constexpr int warpsPerBlock = ThreadsPerBlock / kWarpSize;
        return (V*) &getKeyStart()[warpsPerBlock * HeapSize];
    }

    // Returns a pointer past the end of our block-wide heap storage
    inline __device__ void* getStorageEnd()
    {
        constexpr int warpsPerBlock = ThreadsPerBlock / kWarpSize;
        return getValueStart() + (warpsPerBlock * HeapSize);
    }

    inline __device__ void add(K k, V v)
    {
        warpHeap<K, V, HeapSize, Dir>(k, v, heapHead, heapK, heapV);
    }

    // Reduce all per-warp heaps to a unified, sorted list
    inline __device__ void reduceHeaps()
    {
        constexpr int allHeapSize = (ThreadsPerBlock / kWarpSize) * HeapSize;

        if (Dir)
        {
            bitonicSort<GTComp<K, V>, K, V, allHeapSize, ThreadsPerBlock>(
                getKeyStart(), getValueStart(), GTComp<K, V>());
        }
        else
        {
            bitonicSort<LTComp<K, V>, K, V, allHeapSize, ThreadsPerBlock>(
                getKeyStart(), getValueStart(), LTComp<K, V>());
        }
    }

private:
    void* heapBase;
    K heapHead;
    K* heapK;
    V* heapV;
};

template <typename V, typename IndexType, typename OutIndexType, int ThreadsPerBlock, int HeapSize,
    bool Dir>
__global__ void selectRowsViaHeap(const V* input, // m x n
    V* outKeys,                                   // m x k
    OutIndexType* outIndices,                     // m x k
    V initVal, IndexType initIndex, int m, int n, int k)
{
    extern __shared__ float smem[];

    Heap<V, IndexType, ThreadsPerBlock, HeapSize, Dir> heap(smem, initVal, initIndex);

    auto inputStart = &input[blockIdx.x * n];

    // FIXME choose desired unroll level
    constexpr int Unroll = 1;
    V vals[Unroll];

    for (int i = threadIdx.x; i < n; i += blockDim.x * Unroll)
    {
#ifndef __HIP_PLATFORM_HCC__
#pragma unroll
#endif
        for (int j = 0; j < Unroll; ++j)
        {
            vals[j] = inputStart[i + j * blockDim.x];
        }

#ifndef __HIP_PLATFORM_HCC__
#pragma unroll
#endif
        for (int j = 0; j < Unroll; ++j)
        {
            heap.add(vals[j], (IndexType) i + j * blockDim.x);
        }
    }

    // When finished, we restructure the heaps in shared memory
    // The heaps are actually of size HeapSize - 1 (e.g., 32 -> 31); the
    // extra element should have remained untouched, so we can still
    // sort things in-place as a power of 2.
    __syncthreads();

    heap.reduceHeaps();

    auto outKeysStart = &outKeys[blockIdx.x * k];
    auto outIndicesStart = &outIndices[blockIdx.x * k];

    auto heapK = heap.getKeyStart();
    auto heapV = heap.getValueStart();

    // Write out the final k-selected values; they should be all
    // together
    for (int i = threadIdx.x; i < n && i < k; i += blockDim.x)
    {
        outKeysStart[i] = heapK[i];
        outIndicesStart[i] = (OutIndexType) heapV[i];
    }
}

template <typename T, int kHeapSize, bool kSelectMax = true>
void RunHeapSelectionImpl(
    const T* input, const int outer_size, const int inner_size, const int k, T* values, T* indices, hipStream_t stream)
{
    constexpr int kBlockSize = 256;
    constexpr int kNumWarps = kBlockSize / kWarpSize;
    constexpr int smem = kNumWarps * kHeapSize * (sizeof(T) + sizeof(T));
    constexpr T kInitVal = kSelectMax ? std::numeric_limits<T>::lowest() : std::numeric_limits<T>::max();
    selectRowsViaHeap<T, T, T, kBlockSize, kHeapSize, kSelectMax><<<outer_size, kBlockSize, smem, stream>>>(
        input, values, indices, kInitVal, std::numeric_limits<int64_t>::max(), outer_size, inner_size, k);
}

// template <typename T, bool kSelectMax = true>
// void RunRadixSelectionImpl(
//    const T* input,
//    const int64_t outer_size,
//    const int64_t inner_size,
//    const int k,
//    T* values,
//    int64_t* indices,
//    HIPContext* context) {
//  const int block = std::min(
//      math::RoundUp(static_cast<int>(inner_size), kWarpSize),
//      CAFFE_CUDA_NUM_THREADS);
//  gatherTopK<T, kSelectMax, int64_t>
//  <<<outer_size, block, 0, context->hip_stream()>>>(
//      input, inner_size, k, outer_size, values, indices);
//  // Unfortunately the output is not currently sorted, and there is no batch
//  // sorting utility available. Iterate over all of the slices and sort them
//  // in-place using Thrust.
//  for (int i = 0; i < outer_size; ++i) {
//    thrust::sort_by_key(
//        thrust::cuda::par.on(context->hip_stream()),
//        values + i * k,
//        values + i * k + (k <= inner_size ? k : inner_size),
//        indices + i * k,
//        thrust::greater<T>());
//  }
//}

pluginStatus_t LastDimTopKInference(hipStream_t stream, const int num_threads, void* out_value, void* out_indices,
    const void* in_data, const int K, const int N, const bool is_ascend)
{
    if (is_ascend)
    {
        // If k is small, uses heap selection, otherwise uses radix selection.
        if (K < 32)
        {
            RunHeapSelectionImpl<float, 32, false>(
                (float*) in_data, num_threads, N, K, (float*) out_value, (float*) out_indices, stream);
        }
        else if (K < 128)
        {
            RunHeapSelectionImpl<float, 128, false>(
                (float*) in_data, num_threads, N, K, (float*) out_value, (float*) out_indices, stream);
        }
        else if (K < 512)
        {
            RunHeapSelectionImpl<float, 512, false>(
                (float*) in_data, num_threads, N, K, (float*) out_value, (float*) out_indices, stream);
        }
        else
        {
            //      RunRadixSelectionImpl<float, true>((float *)in_data, num_threads, N,
            //                                              K, (float *)out_value,
            //                                              (float *)out_indices, stream);
        }
    }
    else
    {
        // If k is small, uses heap selection, otherwise uses radix selection.
        if (K < 32)
        {
            RunHeapSelectionImpl<float, 32, true>(
                (float*) in_data, num_threads, N, K, (float*) out_value, (float*) out_indices, stream);
        }
        else if (K < 128)
        {
            RunHeapSelectionImpl<float, 128, true>(
                (float*) in_data, num_threads, N, K, (float*) out_value, (float*) out_indices, stream);
        }
        else if (K < 512)
        {
            RunHeapSelectionImpl<float, 512, true>(
                (float*) in_data, num_threads, N, K, (float*) out_value, (float*) out_indices, stream);
        }
        else
        {
            //      RunRadixSelectionImpl<float, false>((float *)in_data, num_threads, N,
            //                                              K, (float *)out_value,
            //                                              (float *)out_indices, stream);
        }
    }
    return STATUS_SUCCESS;
}