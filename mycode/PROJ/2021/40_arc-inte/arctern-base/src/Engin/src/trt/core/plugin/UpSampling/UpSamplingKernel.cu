#include "hip/hip_runtime.h"
#include "UpSamplingKernel.h"

template <typename DType>
__global__ void UpsamplingKernel(DType* out, const DType* in, const int batchSize, const int input_h, const int input_w,
    const int output_h, const int output_w, const int scale)
{
    int x0 = threadIdx.x + blockIdx.x * blockDim.x;
    int y0 = threadIdx.y + blockIdx.y * blockDim.y;
    int z0 = blockIdx.z;

    int istride = input_w;
    int ostride = output_w;
    int ibatchstride = input_h * istride;
    int obatchstride = output_h * ostride;

    for (int batch = z0; batch < batchSize; batch += gridDim.z)
    {
        for (int oy = y0; oy < output_h; oy += blockDim.y * gridDim.y)
        {
            for (int ox = x0; ox < output_w; ox += blockDim.x * gridDim.x)
            {
                int ix = int(ox / scale);
                int iy = int(oy / scale);
                out[batch * obatchstride + oy * ostride + ox] = in[batch * ibatchstride + iy * istride + ix];
            }
        }
    }
}

pluginStatus_t UpSamplingInference(hipStream_t stream, void* outputs, const void* inputs, int batchSize,
    const int ndim, const int* input_shape, const int* output_shape, const int scale)
{
    const int kMaxGridNum = 65535;
    dim3 block(32, 16);
    dim3 grid((output_shape[3] - 1) / block.x + 1, (output_shape[2] - 1) / block.y + 1,
        std::min(output_shape[0] * output_shape[1] * batchSize, kMaxGridNum));

    UpsamplingKernel<float><<<grid, block, 0, stream>>>((float*) outputs, (float*) inputs,
        output_shape[0] * output_shape[1] * batchSize, input_shape[2], input_shape[3], output_shape[2], output_shape[3],
        scale);
    return STATUS_SUCCESS;
}
