#include "hip/hip_runtime.h"
#include "PadKernel.h"

template <typename DType>
__global__ void image_2d_pad_edge_kernel(DType* dst, const DType* src, const int input_shape_c, const int input_shape_h,
    const int input_shape_w, const int output_shape_c, const int output_shape_h, const int output_shape_w,
    const int padT, const int padL)
{
    int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
    int plane = blockIdx.y;
    int batch = blockIdx.z;
    if (outputPointId >= output_shape_h * output_shape_w)
    {
        return;
    }
    int outputPointX = outputPointId % output_shape_w;
    int outputPointY = outputPointId / output_shape_w;

    int iStartX = max(0, -padL);
    int iStartY = max(0, -padT);
    int oStartX = max(0, padL);
    int oStartY = max(0, padT);

    int inputPointX = min(max(padL, outputPointX), static_cast<int>(input_shape_w) + padL - 1) - oStartX + iStartX;
    int inputPointY = min(max(padT, outputPointY), static_cast<int>(input_shape_h) + padT - 1) - oStartY + iStartY;

    DType valueToCopy = src[batch * input_shape_c * input_shape_h * input_shape_w
        + plane * input_shape_h * input_shape_w + inputPointY * input_shape_w + inputPointX];
    dst[batch * output_shape_c * output_shape_h * output_shape_w + plane * output_shape_h * output_shape_w
        + outputPointY * output_shape_w + outputPointX]
        = valueToCopy;

    //  DType valueToCopy = src[batch][plane][inputPointY][inputPointX];
    //  dst[batch][plane][outputPointY][outputPointX] = valueToCopy;
}

template <typename DType>
inline void image_pad_edge(hipStream_t stream, DType* dst, const DType* src, const int batchSize, const int ndim,
    const int* input_shape, const int* output_shape, const std::vector<int> pad)
{
    const int kBaseThreadBits = 8;
    const int kBaseThreadNum = 1 << kBaseThreadBits;

    if (ndim == 4)
    {
        const int padT = pad[4];
        const int padL = pad[6];
        int xGridSize = (output_shape[2] * output_shape[3] + kBaseThreadNum - 1) / kBaseThreadNum;
        dim3 dimGrid(xGridSize, output_shape[1], output_shape[0] * batchSize);
        image_2d_pad_edge_kernel<DType><<<dimGrid, kBaseThreadNum, 0, stream>>>(dst, src, input_shape[1],
            input_shape[2], input_shape[3], output_shape[1], output_shape[2], output_shape[3], padT, padL);
    }
}

template <typename DType>
__global__ void image_2d_pad_constant_kernel(DType* dst, const DType* src, const int input_shape_c,
    const int input_shape_h, const int input_shape_w, const int output_shape_c, const int output_shape_h,
    const int output_shape_w, const int padT, const int padL, const DType constant)
{
    int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
    int plane = blockIdx.y;
    int batch = blockIdx.z;
    if (outputPointId >= output_shape_h * output_shape_w)
    {
        return;
    }
    // cast sizes to int to use in min/max
    int Ny = input_shape_h;
    int Nx = input_shape_w;

    int outputPointX = outputPointId % output_shape_w;
    int outputPointY = outputPointId / output_shape_w;

    int checkT = max(0, outputPointY - padT + 1);
    int checkB = max(0, padT + Ny - outputPointY);
    int checkL = max(0, outputPointX - padL + 1);
    int checkR = max(0, padL + Nx - outputPointX);

    int inputPointX = min(max(outputPointX - padL, 0), Nx - 1);
    int inputPointY = min(max(outputPointY - padT, 0), Ny - 1);

    // 1 if need padding, 0 if not
    int need_pad = !(checkT * checkB * checkL * checkR);

    DType valueToCopy = src[batch * input_shape_c * input_shape_h * input_shape_w
        + plane * input_shape_h * input_shape_w + inputPointY * input_shape_w + inputPointX];
    dst[batch * output_shape_c * output_shape_h * output_shape_w + plane * output_shape_h * output_shape_w
        + outputPointY * output_shape_w + outputPointX]
        = valueToCopy * (!need_pad) + need_pad * constant;
}

template <typename DType>
inline void image_pad_constant(hipStream_t stream, DType* dst, const DType* src, const int batchSize, const int ndim,
    const int* input_shape, const int* output_shape, const std::vector<int> pad, const float constant)
{
    const int kBaseThreadBits = 8;
    const int kBaseThreadNum = 1 << kBaseThreadBits;

    if (ndim == 4)
    {
        const int padT = pad[4];
        const int padL = pad[6];
        int xGridSize = (output_shape[2] * output_shape[3] + kBaseThreadNum - 1) / kBaseThreadNum;
        dim3 dimGrid(xGridSize, output_shape[1], output_shape[0] * batchSize);
        image_2d_pad_constant_kernel<DType><<<dimGrid, kBaseThreadNum, 0, stream>>>(dst, src, input_shape[1],
            input_shape[2], input_shape[3], output_shape[1], output_shape[2], output_shape[3], padT, padL, constant);
    }
}

pluginStatus_t PadInference(hipStream_t stream, void* outputs, const void* inputs, const int batchSize, const int ndim,
    const int* input_shape, const int* output_shape, const std::vector<int> pad_width, const int mode,
    const float constant_value)
{
    switch (mode)
    {
    case pad_enum::kEdge:
        image_pad_edge<float>(
            stream, (float*) outputs, (float*) inputs, batchSize, ndim, input_shape, output_shape, pad_width);
        break;
    case pad_enum::kConstant:
        image_pad_constant<float>(stream, (float*) outputs, (float*) inputs, batchSize, ndim, input_shape, output_shape,
            pad_width, constant_value);
        break;
        //  case pad_enum::kReflect:
        //    image_pad_reflect<float>(stream, (float *)outputs, (float *)inputs,
        //                             pad_width);
        //    break;
    default: break;
    }
    return STATUS_SUCCESS;
}