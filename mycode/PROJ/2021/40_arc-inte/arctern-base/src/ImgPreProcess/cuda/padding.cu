#include "hip/hip_runtime.h"
//
// Created by Admin on 2021/7/9.
//

#include "padding.cuh"
#include "vec_math.hpp"

namespace arctern{ namespace cuda{ namespace device{


template <typename T>
__global__ void padding_k(const PtrStepSz<T> src, PtrStepSz<T> dst,int top,int left,T borderValue)
{
  const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
  const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dst_x < dst.cols && dst_y < dst.rows)
  {
    /*if(dst_x < left || dst_y < top){
      dst(dst_y, dst_x) = borderValue;
    }
    else if(dst_x >= left + src.cols || dst_y >= top + src.rows){
      dst(dst_y, dst_x) = borderValue;
    } else{
      int src_x = dst_x - left;
      int src_y = dst_y - top;
      dst(dst_y,dst_x) = src(src_y,src_x);
    }*/

    if(dst_y >= top && dst_x >= left && dst_y < src.rows + top && dst_x < src.cols + left){
      int src_x = dst_x - left;
      int src_y = dst_y - top;
      dst(dst_y,dst_x) = src(src_y,src_x);
    } else{
      dst(dst_y,dst_x) = borderValue;
    }
  }
}

template <typename T>
void padding(const PtrStepSz<T>& src,const PtrStepSz<T>& dst,int top,int left,T borderValue,hipStream_t *m_pStream)
{
  const dim3 block(32, 8);
  const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  padding_k<<<grid, block, 0, *m_pStream>>>(src, dst, top, left, borderValue);

  cudaSafeCall( hipGetLastError() );

  //cudaSafeCall(hipStreamSynchronize(m_pStream));
}

template <typename T>
void padding(const PtrStepSzb& src, PtrStepSzb& dst,int top,int left, T borderValue,hipStream_t *m_pStream)
{
  padding<T>(static_cast<PtrStepSz<T> >(src), static_cast<PtrStepSz<T> > (dst), top, left, borderValue, m_pStream);
}

template void padding<uchar>(const PtrStepSzb& src, PtrStepSzb& dst,int top,int left, uchar borderValue,hipStream_t *m_pStream);
template void padding<float>(const PtrStepSzb& src, PtrStepSzb& dst,int top,int left, float borderValue,hipStream_t *m_pStream);

template void padding<uchar3>(const PtrStepSzb& src, PtrStepSzb& dst,int top,int left, uchar3 borderValue,hipStream_t *m_pStream);
template void padding<float3>(const PtrStepSzb& src, PtrStepSzb& dst,int top,int left, float3 borderValue,hipStream_t *m_pStream);

}}}