#include "hip/hip_runtime.h"
//
// Created by Admin on 2021/7/5.
//

#include "normalized.cuh"
#include "vec_math.hpp"

namespace arctern{ namespace cuda{ namespace device{
//binary operators (vecA *+ vecB)
#define ARCTERN_CUDA_OP(op,type) \
    __device__ __forceinline__ float operator op(const type &a,const float4 &b) { \
        return a op b.x; \
    }\
    __device__ __forceinline__ float ## 3 operator op(const type ## 3 &a,const float4 &b) { \
        return VecTraits<float ## 3>::make(a.x op b.x, a.y op b.y, a.z op b.z); \
    }\
    __device__ __forceinline__ float ## 4 operator op(const type ## 4 &a,const float4 &b) { \
        return VecTraits<float ## 4>::make(a.x op b.x, a.y op b.y, a.z op b.z, a.w op b.w); \
    }

ARCTERN_CUDA_OP(*,uchar)
ARCTERN_CUDA_OP(*,float)
ARCTERN_CUDA_OP(+,uchar)
ARCTERN_CUDA_OP(+,float)


template <typename T,typename U>
__global__ void normalized_k(const PtrStepSz<T> src, PtrStepSz<U> dst, const float4 alpha, const float4 beta)
{
  const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
  const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dst_x < dst.cols && dst_y < dst.rows)
  {
    const float src_x = dst_x ;
    const float src_y = dst_y ;

    dst(dst_y, dst_x) = src(src_y, src_x) * alpha + beta;
  }
}

template <typename T,typename U>
void normalized(const PtrStepSz<T>& src,const PtrStepSz<U>& dst, const float4 alpha, const float4 beta,hipStream_t *m_pStream)
{
  const dim3 block(64, 16);
  const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  normalized_k<<<grid, block,0, *m_pStream>>>(src, dst, alpha, beta);

  cudaSafeCall( hipGetLastError() );

  //cudaSafeCall(hipStreamSynchronize(m_pStream));
}

template <typename T,typename U>
void normalized(const PtrStepSzb& src, PtrStepSzb& dst, const float4 alpha, const float4 beta,hipStream_t *m_pStream)
{
  normalized<T,U>(static_cast<PtrStepSz<T> >(src), static_cast<PtrStepSz<U> > (dst),  alpha,  beta ,m_pStream);
}

template void normalized<uchar,float>(const PtrStepSzb& src, PtrStepSzb& dst, const float4 alpha, const float4 beta,hipStream_t *m_pStream);
template void normalized<float,float>(const PtrStepSzb& src, PtrStepSzb& dst, const float4 alpha, const float4 beta,hipStream_t *m_pStream);

template void normalized<uchar3,float3>(const PtrStepSzb& src, PtrStepSzb& dst, const float4 alpha, const float4 beta,hipStream_t *m_pStream);
template void normalized<float3,float3>(const PtrStepSzb& src, PtrStepSzb& dst, const float4 alpha, const float4 beta,hipStream_t *m_pStream);
}}}