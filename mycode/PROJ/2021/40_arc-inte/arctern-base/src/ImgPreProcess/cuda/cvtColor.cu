#include "hip/hip_runtime.h"
//
// Created by Admin on 2021/7/7.
//

#include "cvtColor.cuh"
#include "common.hpp"
#include "util.hpp"
#include "vec_math.hpp"
namespace arctern {
namespace cuda {
namespace device {

template<typename T,typename U>
__global__ void cvtRGB2GRAY_k(const PtrStepSz<T> src, PtrStepSz<U> dst)
{
  const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
  const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dst_x < dst.cols && dst_y < dst.rows) {
    int src_x = dst_x;
    int src_y = dst_y;
    dst(dst_y,dst_x) = 0.299*src(src_y,src_x).x + 0.587*src(src_y,src_x).y + 0.114*src(src_y,src_x).z;
  }
}

template<typename T,typename U>
void cvtRGB2GRAY(const PtrStepSz<T> &src, PtrStepSz<U> dst,hipStream_t *m_pStream) {
  const dim3 block(64, 16);
  const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  cvtRGB2GRAY_k<T,U><<<grid, block,0,*m_pStream>>>(src, dst);

  cudaSafeCall(hipGetLastError());

  //cudaSafeCall(hipStreamSynchronize(m_pStream));
}

template<typename T,typename U>
void cvtRGB2GRAY(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream) {
  cvtRGB2GRAY<T,U>(static_cast<PtrStepSz<T> >(src), static_cast<PtrStepSz<U> > (dst),m_pStream);
}

template void cvtRGB2GRAY<uchar3,uchar>(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream);
template void cvtRGB2GRAY<float3,float>(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream);


template<typename T,typename U>
__global__ void cvtBGR2GRAY_k(const PtrStepSz<T> src, PtrStepSz<U> dst)
{
  const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
  const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dst_x < dst.cols && dst_y < dst.rows) {
    int src_x = dst_x;
    int src_y = dst_y;
    dst(dst_y,dst_x) = 0.114*src(src_y,src_x).x + 0.587*src(src_y,src_x).y + 0.299*src(src_y,src_x).z;
  }
}

template<typename T,typename U>
void cvtBGR2GRAY(const PtrStepSz<T> &src, PtrStepSz<U> dst,hipStream_t *m_pStream) {
  const dim3 block(64, 16);
  const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  cvtBGR2GRAY_k<T,U><<<grid, block,0,*m_pStream>>>(src, dst);

  cudaSafeCall(hipGetLastError());

  //cudaSafeCall(hipStreamSynchronize(m_pStream));
}

template<typename T,typename U>
void cvtBGR2GRAY(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream) {
  cvtBGR2GRAY<T,U>(static_cast<PtrStepSz<T> >(src), static_cast<PtrStepSz<U> > (dst),m_pStream);
}

template void cvtBGR2GRAY<uchar3,uchar>(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream);
template void cvtBGR2GRAY<float3,float>(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream);


template<typename T>
__global__ void cvtBGR2RGB_k(const PtrStepSz<T> src, PtrStepSz<T> dst)
{
  const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
  const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dst_x < dst.cols && dst_y < dst.rows) {
    int src_x = dst_x;
    int src_y = dst_y;
    dst(dst_y,dst_x).x = src(src_y,src_x).z;
    dst(dst_y,dst_x).y = src(src_y,src_x).y;
    dst(dst_y,dst_x).z = src(src_y,src_x).x;
  }
}

template<typename T>
void cvtBGR2RGB(const PtrStepSz<T> &src, PtrStepSz<T> dst,hipStream_t *m_pStream) {
  const dim3 block(64, 16);
  const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  cvtBGR2RGB_k<T> <<< grid, block,0 ,*m_pStream >>>(src, dst);

  cudaSafeCall(hipGetLastError());

  //cudaSafeCall(hipStreamSynchronize(m_pStream));
}

template<typename T>
void cvtBGR2RGB(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream) {
  cvtBGR2RGB<T>(static_cast<PtrStepSz<T> >(src), static_cast<PtrStepSz<T> > (dst),m_pStream);
}

template void cvtBGR2RGB<uchar3>(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream);
template void cvtBGR2RGB<float3>(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream);


template<typename T,typename U>
__global__ void ExtractChannel_k(const PtrStepSz<T> src, PtrStepSz<T> dst)
{
  const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
  const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dst_x < dst.cols && dst_y < dst.rows) {
    U *px = (U*)(((uchar*)dst.ptr()) + (0 * dst.cols * dst.rows  + dst_y * dst.cols + dst_x)*sizeof(U) );
    U *py = (U*)(((uchar*)dst.ptr()) + (1 * dst.cols * dst.rows + dst_y * dst.cols + dst_x)*sizeof(U) );
    U *pz = (U*)(((uchar*)dst.ptr()) + (2 * dst.cols * dst.rows + dst_y * dst.cols + dst_x)*sizeof(U) );

    *px = src(dst_y,dst_x).x;
    *py = src(dst_y,dst_x).y;
    *pz = src(dst_y,dst_x).z;
  }
}

template<typename T,typename U>
void ExtractChannel(const PtrStepSz<T> &src, PtrStepSz<T> dst,hipStream_t *m_pStream) {
  const dim3 block(64, 16);
  const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  ExtractChannel_k<T,U><<<grid, block, 0 ,*m_pStream>>>(src, dst);

  cudaSafeCall(hipGetLastError());

  //cudaSafeCall(hipStreamSynchronize(m_pStream));
}

template<typename T,typename U>
void ExtractChannel(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream) {
  ExtractChannel<T,U>(static_cast<PtrStepSz<T> >(src), static_cast<PtrStepSz<T> > (dst),m_pStream);
}

template void ExtractChannel<uchar3,uchar>(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream);
template void ExtractChannel<float3,float>(const PtrStepSzb &src, PtrStepSzb &dst,hipStream_t *m_pStream);

template<typename T, typename U>
__global__ void cvt3CTo4C_k(const PtrStepSz<T> src, PtrStepSz<U> dst) {
  const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
  const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dst_x < dst.cols && dst_y < dst.rows) {
    const float src_x = dst_x;
    const float src_y = dst_y;

    dst(dst_y, dst_x).x = src(src_y, src_x).x;
    dst(dst_y, dst_x).y = src(src_y, src_x).y;
    dst(dst_y, dst_x).z = src(src_y, src_x).z;
    dst(dst_y, dst_x).w = 255;
  }
}


template<typename T, typename U>
void cvt3CTo4C(const PtrStepSz<T> &src, const PtrStepSz<U> &dst) {
  const dim3 block(32, 8);
  const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  cvt3CTo4C_k<<<grid, block>>>(src, dst);

  cudaSafeCall(hipGetLastError());

  cudaSafeCall(hipDeviceSynchronize());
}

template<typename T, typename U>
void cvt3CTo4C(const PtrStepSzb &src, PtrStepSzb &dst) {
  cvt3CTo4C<T, U>(static_cast<PtrStepSz<T> >(src), static_cast<PtrStepSz<U> > (dst));
}

template void cvt3CTo4C<uchar3, uchar4>(const PtrStepSzb &src, PtrStepSzb &dst);
template void cvt3CTo4C<float3, float4>(const PtrStepSzb &src, PtrStepSzb &dst);


template<typename T, typename U>
__global__ void cvt4CTo3C_k(const PtrStepSz<T> src, PtrStepSz<U> dst) {
  const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
  const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dst_x < dst.cols && dst_y < dst.rows) {
    const float src_x = dst_x;
    const float src_y = dst_y;

    dst(dst_y, dst_x).x = src(src_y, src_x).x;
    dst(dst_y, dst_x).y = src(src_y, src_x).y;
    dst(dst_y, dst_x).z = src(src_y, src_x).z;
  }
}

template<typename T, typename U>
void cvt4CTo3C(const PtrStepSz<T> &src, const PtrStepSz<U> &dst) {
  const dim3 block(32, 8);
  const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  cvt4CTo3C_k<<<grid, block>>>(src, dst);

  cudaSafeCall(hipGetLastError());

  cudaSafeCall(hipDeviceSynchronize());
}

template<typename T, typename U>
void cvt4CTo3C(const PtrStepSzb &src, PtrStepSzb &dst) {
  cvt4CTo3C<T, U>(static_cast<PtrStepSz<T> >(src), static_cast<PtrStepSz<U> > (dst));
}

template void cvt4CTo3C<uchar4, uchar3>(const PtrStepSzb &src, PtrStepSzb &dst);
template void cvt4CTo3C<float4, float3>(const PtrStepSzb &src, PtrStepSzb &dst);

}
}
}