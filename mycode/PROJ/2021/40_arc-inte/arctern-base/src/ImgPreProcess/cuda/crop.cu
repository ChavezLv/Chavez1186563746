#include "hip/hip_runtime.h"
//
// Created by Admin on 2021/11/1.
//

#include "crop.cuh"
namespace arctern{ namespace cuda{ namespace device{


template <typename T>
__global__ void crop_k(const PtrStepSz<T> src, PtrStepSz<T> dst, ArcternRect rROI)
{
  const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
  const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dst_x < dst.cols && dst_y < dst.rows)
  {
      int src_x = dst_x + (int)rROI.x;
      int src_y = dst_y + (int)rROI.y;
      dst(dst_y,dst_x) = src(src_y,src_x);
  }
}

template <typename T>
void crop(const PtrStepSz<T>& src,const PtrStepSz<T>& dst, ArcternRect rROI,hipStream_t *m_pStream)
{
  const dim3 block(32, 8);
  const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  crop_k<<<grid, block,0,*m_pStream>>>(src, dst, rROI);

  cudaSafeCall( hipGetLastError() );

  //cudaSafeCall(hipStreamSynchronize(m_pStream));
}

template <typename T>
void crop(const PtrStepSzb& src, PtrStepSzb& dst, ArcternRect rROI,hipStream_t *m_pStream)
{

  crop<T>(static_cast<PtrStepSz<T> >(src), static_cast<PtrStepSz<T> > (dst), rROI,m_pStream);
}

template void crop<uchar>(const PtrStepSzb& src, PtrStepSzb& dst, ArcternRect rROI,hipStream_t *m_pStream);
template void crop<float>(const PtrStepSzb& src, PtrStepSzb& dst, ArcternRect rROI,hipStream_t *m_pStream);

template void crop<uchar3>(const PtrStepSzb& src, PtrStepSzb& dst, ArcternRect rROI,hipStream_t *m_pStream);
template void crop<float3>(const PtrStepSzb& src, PtrStepSzb& dst, ArcternRect rROI,hipStream_t *m_pStream);

}}}